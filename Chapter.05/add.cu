
	
#include <hip/hip_runtime.h>
#include <stdio.h>

	// CUDA kernel function to add two numbers
	__global__ void add(int *a, int *b, int *c) {
	    *c = *a + *b;
	printf("GPU: computed %d + %d = %d\n", *a, *b, *c);
	}

	int main(void) {
		int a, b, c;	            // host copies of a, b, c
		int *d_a, *d_b, *d_c;	     // device copies of a, b, c
		int size = sizeof(int);
		
		// Allocate space for device copies of a, b, c
		hipMalloc((void **)&d_a, size);
		hipMalloc((void **)&d_b, size);
		hipMalloc((void **)&d_c, size);
		// Setup input values
		a = 2;
		b = 7;

		// Copy inputs to device
		hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
		// Launch add() kernel on GPU
		add<<<1,1>>>(d_a, d_b, d_c);

		// Copy result back to host
		hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

		// Print the result on the host
		printf("CPU: received result %d + %d = %d\n", a, b, c);

		// Cleanup
		hipFree(d_a); 
		hipFree(d_b); 
		hipFree(d_c);

		return 0;
	}

